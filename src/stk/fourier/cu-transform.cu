#include "hip/hip_runtime.h"
#include <stk/cuda/tools.hpp>

#define PI 3.141592653589793238462643

__global__ void stk_cuKrnl_fourierTransform(
	const float2* i_srcPos, const float2* i_srcVal, int i_srcSize,
	const float2* i_dstPos, float2* o_dstVal, int i_dstSize,
	float i_dir, float i_normalization)
{
	int i;
	int u = (blockIdx.x * blockDim.x + threadIdx.x);
	float ph;
	float2 res;
	res.x = 0;
	res.y = 0;
	
	if(u<i_dstSize)
	{
		for(i=0; i<i_srcSize; i++)
		{
			ph = i_srcPos[i].x*i_dstPos[u].x + i_srcPos[i].y*i_dstPos[u].y;
			ph *= i_dir*2.0*PI;
			
			res.x += i_srcVal[i].x * cos(ph) - i_srcVal[i].y * sin(ph);
			res.y += i_srcVal[i].y * cos(ph) + i_srcVal[i].x * sin(ph);
		}
		
		o_dstVal[u].x = res.x / i_normalization;
		o_dstVal[u].y = res.y / i_normalization;
	}
}

void stk_cuFourierTransform(
	const float* i_srcPos, const float* i_srcVal, int i_srcSize,
	const float* i_dstPos, float* o_dstVal, int i_dstSize,
	float i_dir, float i_normalization)
{	
	dim3 dimGrid, dimBlock;
	stk_cuGetSizes(dimGrid, dimBlock, i_srcSize, i_dstSize);

	/* INIT ***********************************************************/
	const int srcByteSz = i_srcSize*sizeof(float2);
	const int destByteSz = i_dstSize*sizeof(float2);
	
	float2* dvcSrcPos;
	hipMalloc((void**) &dvcSrcPos, srcByteSz);
	float2* dvcSrcVal;
	hipMalloc((void**) &dvcSrcVal, srcByteSz);
	float2* dvcDestPos;
	hipMalloc((void**) &dvcDestPos, destByteSz);
	float2* dvcDestVal;
	hipMalloc((void**) &dvcDestVal, destByteSz);
	
	/* START ***********************************************************/
	hipMemcpy(dvcSrcPos, i_srcPos, srcByteSz, hipMemcpyHostToDevice);
	hipMemcpy(dvcSrcVal, i_srcVal, srcByteSz, hipMemcpyHostToDevice);
	hipMemcpy(dvcDestPos, i_dstPos, destByteSz, hipMemcpyHostToDevice);
	
	stk_cuKrnl_fourierTransform <<< dimGrid, dimBlock >>> (
		dvcSrcPos, dvcSrcVal, i_srcSize,
		dvcDestPos, dvcDestVal, i_dstSize,
		i_dir, i_normalization);
		
	hipMemcpy(o_dstVal, dvcDestVal, destByteSz, hipMemcpyDeviceToHost);
	
	hipFree(dvcSrcPos);
	hipFree(dvcSrcVal);
	hipFree(dvcDestPos);
	hipFree(dvcDestVal);
}
